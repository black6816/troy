#include "hip/hip_runtime.h"
#include "kernelutils.cuh"

#define KERNEL_CALL(funcname, n) size_t block_count = ceilDiv_(n, 256); funcname<<<block_count, 256>>>
#define POLY_ARRAY_ARGCALL poly_size, coeff_modulus_size, poly_modulus_degree
#define GET_INDEX size_t gindex = blockDim.x * blockIdx.x + threadIdx.x
#define GET_INDEX_COND_RETURN(n) size_t gindex = blockDim.x * blockIdx.x + threadIdx.x; if (gindex >= (n)) return
#define FOR_N(name, count) for (size_t name = 0; name < count; name++)

namespace troy {

    namespace kernel_util {

        // using troy::util::ConstDevicePointer;
        // using troy::util::DevicePointer;
        // using CPointer = ConstDevicePointer<uint64_t>;
        // using Pointer = DevicePointer<uint64_t>;
        // using MPointer = ConstDevicePointer<Modulus>;
        // using troy::util::MultiplyUIntModOperand;
        // using uint128_t = unsigned __int128;






        


        __global__ void gAddPolyCoeffmod(
            const uint64_t* operand1,
            const uint64_t* operand2,
            POLY_ARRAY_ARGUMENTS,
            const Modulus* modulus,
            uint64_t* result
        ) {
            GET_INDEX_COND_RETURN(poly_modulus_degree);
            FOR_N(rns_index, coeff_modulus_size) {
                const uint64_t modulusValue = DeviceHelper::getModulusValue(modulus[rns_index]);
                FOR_N(poly_index, poly_size) {
                    size_t id = (poly_index * coeff_modulus_size + rns_index) * poly_modulus_degree + gindex;
                    uint64_t sum = operand1[id] + operand2[id];
                    result[id] = sum >= modulusValue ? sum - modulusValue : sum;
                }
            }
        }

        void kAddPolyCoeffmod(
            CPointer operand1,
            CPointer operand2,
            POLY_ARRAY_ARGUMENTS,
            MPointer modulus,
            Pointer result
        ) {
            KERNEL_CALL(gAddPolyCoeffmod, poly_modulus_degree)(
                operand1.get(), operand2.get(),
                POLY_ARRAY_ARGCALL, modulus.get(), result.get()
            );
        }

        __device__ inline uint64_t dDyadicSingle(uint64_t o1, uint64_t o2, uint64_t modulus_value, uint64_t const_ratio_0, uint64_t const_ratio_1) {
            
            uint64_t z[2], tmp1, tmp2[2], tmp3, carry;
            
            // Reduces z using base 2^64 Barrett reduction
            dMultiplyUint64(o1, o2, z);

            // Multiply input and const_ratio
            // Round 1
            dMultiplyUint64HW64(z[0], const_ratio_0, &carry);
            dMultiplyUint64(z[0], const_ratio_1, tmp2);
            tmp3 = tmp2[1] + dAddUint64(tmp2[0], carry, &tmp1);

            // Round 2
            dMultiplyUint64(z[1], const_ratio_0, tmp2);
            carry = tmp2[1] + dAddUint64(tmp1, tmp2[0], &tmp1);

            // This is all we care about
            tmp1 = z[1] * const_ratio_1 + tmp3 + carry;

            // Barrett subtraction
            tmp3 = z[0] - tmp1 * modulus_value;

            // Claim: One more subtraction is enough
            uint64_t sum = ((tmp3 >= modulus_value) ? (tmp3 - modulus_value) : (tmp3));
            uint64_t res = sum >= modulus_value ? sum-modulus_value : sum;
            return res;
        }

        __global__ void gDyadicConvolutionCoeffmod(
            const uint64_t* operand1,
            const uint64_t* operand2_reversed,
            POLY_ARRAY_ARGUMENTS,
            const Modulus* moduli,
            uint64_t* single_poly_result_accumulator
        ) {
            GET_INDEX_COND_RETURN(poly_modulus_degree);
            FOR_N(rns_index, coeff_modulus_size) {
                const uint64_t modulus_value = DeviceHelper::getModulusValue(moduli[rns_index]);
                const uint64_t const_ratio_0 = DeviceHelper::getModulusConstRatio(moduli[rns_index])[0];
                const uint64_t const_ratio_1 = DeviceHelper::getModulusConstRatio(moduli[rns_index])[1];
                FOR_N(poly_index, poly_size) {

                    const uint64_t* o1 = operand1 
                        + (poly_index * coeff_modulus_size + rns_index) * poly_modulus_degree + gindex;
                    const uint64_t* o2 = operand2_reversed - poly_index * coeff_modulus_size * poly_modulus_degree
                        + rns_index * poly_modulus_degree + gindex;
                    uint64_t* res = single_poly_result_accumulator
                        + rns_index * poly_modulus_degree + gindex;
                    
                    // Claim: One more subtraction is enough
                    uint64_t sum = *res + dDyadicSingle(*o1, *o2, modulus_value, const_ratio_0, const_ratio_1);
                    *res = sum >= modulus_value ? sum-modulus_value : sum;
                }
            }
        }

        void kDyadicConvolutionCoeffmod(
            CPointer operand1,
            CPointer operand2_reversed,
            POLY_ARRAY_ARGUMENTS,
            MPointer moduli,
            Pointer single_poly_result_accumulator
        ) {
            KERNEL_CALL(gDyadicConvolutionCoeffmod, poly_modulus_degree) (
                operand1.get(), 
                operand2_reversed.get(), 
                POLY_ARRAY_ARGCALL,
                moduli.get(), single_poly_result_accumulator.get()
            );
        }

        __global__ void gDyadicSquareCoeffmod(
            uint64_t* operand,
            size_t coeff_modulus_size,
            size_t poly_modulus_degree,
            const Modulus* moduli
        ) {
            GET_INDEX_COND_RETURN(poly_modulus_degree);
            size_t d = coeff_modulus_size * poly_modulus_degree;
            FOR_N(rns_index, coeff_modulus_size) {
                const uint64_t modulus_value = DeviceHelper::getModulusValue(moduli[rns_index]);
                const uint64_t const_ratio_0 = DeviceHelper::getModulusConstRatio(moduli[rns_index])[0];
                const uint64_t const_ratio_1 = DeviceHelper::getModulusConstRatio(moduli[rns_index])[1];
                size_t id = rns_index * poly_modulus_degree + gindex;
                operand[2 * d + id] = dDyadicSingle(operand[1 * d + id], operand[1 * d + id], modulus_value, const_ratio_0, const_ratio_1);
                uint64_t cross = dDyadicSingle(operand[0 * d + id], operand[1 * d + id], modulus_value, const_ratio_0, const_ratio_1);
                cross += cross;
                operand[1 * d + id] = cross >= modulus_value ? cross-modulus_value : cross;
                operand[0 * d + id] = dDyadicSingle(operand[0 * d + id], operand[0 * d + id], modulus_value, const_ratio_0, const_ratio_1);
            }
        }

        void kDyadicSquareCoeffmod(
            Pointer operand,
            size_t coeff_modulus_size,
            size_t poly_modulus_degree,
            MPointer moduli
        ) {
            KERNEL_CALL(gDyadicSquareCoeffmod, poly_modulus_degree)(
                operand.get(), coeff_modulus_size, poly_modulus_degree,
                moduli.get()
            );
        }


        __global__ void gMultiplyPolyScalarCoeffmod(
            const uint64_t* poly_array,
            POLY_ARRAY_ARGUMENTS,
            const MultiplyUIntModOperand* reduced_scalar,
            const Modulus* modulus,
            uint64_t* result)
        {
            GET_INDEX_COND_RETURN(poly_modulus_degree);
            FOR_N(rns_index, coeff_modulus_size) {
                FOR_N(poly_index, poly_size) {
                    size_t id = (poly_index * coeff_modulus_size + rns_index) * poly_modulus_degree;
                    result[id] = dMultiplyUintMod(poly_array[id], reduced_scalar[rns_index], modulus[rns_index]);
                }
            }
        }


        void kMultiplyPolyScalarCoeffmod(CPointer poly_array, POLY_ARRAY_ARGUMENTS, uint64_t scalar, MPointer modulus, Pointer result)
        {
            util::DeviceArray<MultiplyUIntModOperand> reduced_scalar(coeff_modulus_size);
            assert(coeff_modulus_size <= 256);
            gSetMultiplyUIntModOperand<<<1, coeff_modulus_size>>>(scalar, modulus.get(), coeff_modulus_size, reduced_scalar.get());
            KERNEL_CALL(gMultiplyPolyScalarCoeffmod, poly_modulus_degree)(
                poly_array.get(), POLY_ARRAY_ARGCALL, reduced_scalar.get(), 
                modulus.get(), result.get()
            ); 
        }

        __global__ void gNegatePolyCoeffmod(
            const uint64_t* poly_array,
            POLY_ARRAY_ARGUMENTS,
            const Modulus* modulus,
            uint64_t *result
        ) {
            GET_INDEX_COND_RETURN(poly_modulus_degree);
            FOR_N(rns_index, coeff_modulus_size) {
                auto modulus_value = DeviceHelper::getModulusValue(modulus[rns_index]);
                FOR_N(poly_index, poly_size) {
                    size_t id = (poly_index * coeff_modulus_size + rns_index) * poly_modulus_degree + gindex;
                    uint64_t coeff = poly_array[id];
                    int64_t non_zero = (coeff != 0);
                    result[id] = (modulus_value - coeff) & static_cast<uint64_t>(-non_zero);
                }
            }
        }

        void kNegatePolyCoeffmod(
            CPointer poly_array, POLY_ARRAY_ARGUMENTS,
            MPointer modulus, Pointer result
        ) {
            KERNEL_CALL(gNegatePolyCoeffmod, poly_modulus_degree)(
                poly_array.get(),
                POLY_ARRAY_ARGCALL,
                modulus.get(),
                result.get()
            );
        }

        __global__ void gSetMultiplyUIntModOperand(uint64_t scalar, const Modulus* moduli, size_t n, MultiplyUIntModOperand* result) {
            GET_INDEX_COND_RETURN(n);
            uint64_t reduced = dBarrettReduce64(scalar, moduli[gindex]);
            result[gindex].operand = reduced;
            std::uint64_t wide_quotient[2]{ 0, 0 };
            std::uint64_t wide_coeff[2]{ 0, result[gindex].operand };
            dDivideUint128Inplace(wide_coeff, DeviceHelper::getModulusValue(moduli[gindex]), wide_quotient);
            result[gindex].quotient = wide_quotient[0];
        }

        
        void kSetPolyArray(
            CPointer poly, POLY_ARRAY_ARGUMENTS, Pointer result
        ) {
            KernelProvider::copyOnDevice(
                result.get(), poly.get(), 
                poly_size * coeff_modulus_size * poly_modulus_degree
            );
        }

        __global__ void gSubPolyCoeffmod(
            const uint64_t* operand1,
            const uint64_t* operand2,
            POLY_ARRAY_ARGUMENTS,
            const Modulus* modulus,
            uint64_t* result
        ) {
            GET_INDEX_COND_RETURN(poly_modulus_degree);
            FOR_N(rns_index, coeff_modulus_size) {
                const uint64_t modulusValue = DeviceHelper::getModulusValue(modulus[rns_index]);
                FOR_N(poly_index, poly_size) {
                    size_t id = (poly_index * coeff_modulus_size + rns_index) * poly_modulus_degree + gindex;
                    uint64_t temp_result;
                    int64_t borrow = dSubUint64(operand1[id], operand2[id], &temp_result);
                    result[id] = temp_result + (modulusValue & static_cast<std::uint64_t>(-borrow));
                }
            }
        }

        void kSubPolyCoeffmod(
            CPointer operand1,
            CPointer operand2,
            POLY_ARRAY_ARGUMENTS,
            MPointer modulus,
            Pointer result
        ) {
            KERNEL_CALL(gSubPolyCoeffmod, poly_modulus_degree)(
                operand1.get(), operand2.get(),
                POLY_ARRAY_ARGCALL, modulus.get(), result.get()
            );
        }


    }
}