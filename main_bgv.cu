#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <iomanip>

#include "src/troy_cpu.h"
#include "src/troy_cuda.cuh"

using namespace troy;
using namespace troy::util;
using std::vector;
using std::string;

#define ASSERT_TRUE(p) if (!(p)) std::cout << "===== Assert failed: line " << std::dec << __LINE__ << "\n"; \
    else std::cout << "ooooo Assert accept: line " << std::dec << __LINE__ << std::endl;
#define ASSERT_FALSE(p) if ((p)) std::cout << "===== Assert failed: line " << std::dec << __LINE__ << "\n"; \
    else std::cout << "ooooo Assert accept: line " << std::dec << __LINE__ << std::endl;
#define ASSERT_EQ(a, b) ASSERT_TRUE((a)==(b))

void printVector(const vector<int64_t>& r, bool full = false) {
    std::cout << "[";
    for (size_t i = 0; i < r.size(); i++) {
        if (r.size() > 8 && !full && i == 4) {
            std::cout << " ...";
            i = r.size() - 4;
        }
        if (i!=0) std::cout << ", ";
        std::cout << r[i];
    }
    std::cout << "]" << std::endl;
}

void printVector(const HostArray<uint64_t>& r, bool full = false) {
    std::cout << "[";
    for (size_t i = 0; i < r.size(); i++) {
        if (r.size() > 8 && !full && i == 4) {
            std::cout << " ...";
            i = r.size() - 4;
        }
        if (i!=0) std::cout << ", ";
        std::cout << r[i];
    }
    std::cout << "]" << std::endl;
}


void printVector(const uint64_t* r, size_t size, bool full = false) {
    std::cout << "[";
    for (size_t i = 0; i < size; i++) {
        if (size > 8 && !full && i == 4) {
            std::cout << " ...";
            i = size - 4;
        }
        if (i!=0) std::cout << ", ";
        std::cout << r[i];
    }
    std::cout << "]" << std::endl;
}

template <typename T>
void printVectorT(const T* r, size_t size, bool full = false) {
    std::cout << "[";
    for (size_t i = 0; i < size; i++) {
        if (size > 8 && !full && i == 4) {
            std::cout << " ...";
            i = size - 4;
        }
        if (i!=0) std::cout << ", ";
        std::cout << r[i];
    }
    std::cout << "]" << std::endl;
}

template<typename T>
vector<T> addVector(const vector<T>& a, const vector<T>& b) {
    assert(a.size() == b.size());
    vector<T> ret(a.size());
    for (size_t i = 0; i < a.size(); i++) 
        ret[i] = a[i] + b[i];
    return ret;
}

template<typename T>
vector<T> negateVector(const vector<T>& a) {
    vector<T> ret(a.size());
    for (size_t i = 0; i < a.size(); i++) 
        ret[i] = -a[i];
    return ret;
}

template<typename T>
vector<T> multiplyVector(const vector<T>& a, const vector<T>& b) {
    assert(a.size() == b.size());
    vector<T> ret(a.size());
    for (size_t i = 0; i < a.size(); i++) 
        ret[i] = a[i] * b[i];
    return ret;
}

size_t ceilDiv(size_t a, size_t b) {
    if (a%b) return (a/b+1);
    return a/b;
}

vector<int64_t> randomVector(size_t count, int data_bound) {
    vector<int64_t> input(count, 0.0);
    for (size_t i = 0; i < count; i++)
    {
        input[i] = rand() % data_bound;
    }
    return input;
}

Ciphertext encrypt(SEALContext& context, BatchEncoder& encoder, Encryptor& encryptor, const vector<int64_t>& message) {
    Plaintext plaintext;
    encoder.encode(message, plaintext);
    Ciphertext ciphertext;
    encryptor.encrypt(plaintext, ciphertext);
    return ciphertext;
}

CiphertextCuda encryptCuda(SEALContext& context, BatchEncoder& encoder, Encryptor& encryptor, const vector<int64_t>& message) {
    return CiphertextCuda(encrypt(context, encoder, encryptor, message));
}

vector<int64_t> decrypt(BatchEncoder& encoder, Decryptor& decryptor, const Ciphertext& ciphertext, size_t slots) {
    Plaintext plaintext;
    decryptor.decrypt(ciphertext, plaintext);
    vector<int64_t> ret(slots);
    encoder.decode(plaintext, ret);
    return ret;
}  

vector<int64_t> decryptCuda(BatchEncoder& encoder, Decryptor& decryptor, const CiphertextCuda& ciphertext, size_t slots) {
    return decrypt(encoder, decryptor, ciphertext.cpu(), slots);
}

__global__ void printNTTTables(const NTTTablesCuda* c) {
    size_t id = threadIdx.x;
    uint64_t p = DeviceHelper::getModulusValue(c[id].modulus());
    printf("%llu\n", p);
}

#define RANDOM_MESSAGE randomVector(slot_size, data_bound)
#define ENCRYPT(msg) encryptCuda(context, encoder, encryptor, msg)
#define DECRYPT(cipher) decryptCuda(encoder, decryptor, cipher, slot_size)

void test_ckks() {
    
    EncryptionParameters parms(SchemeType::bgv);        
    Modulus plain_modulus(PlainModulus::Batching(64, 20));
    parms.setPolyModulusDegree(64);
    parms.setPlainModulus(plain_modulus);
    parms.setCoeffModulus(CoeffModulus::Create(64, { 30, 30, 30 }));

    SEALContext context(parms, false, SecurityLevel::none);
    KeyGenerator keygen(context);
    PublicKey pk;
    keygen.createPublicKey(pk);

    BatchEncoder encoder(context);
    Encryptor encryptor(context, pk);
    Decryptor decryptor(context, keygen.secretKey());

    size_t slot_size = encoder.slotCount();

    Evaluator evaluator(context);

    SEALContextCuda c_context(context);
    EvaluatorCuda c_evaluator(c_context);

    int data_bound = (1 << 4);

    if (false) { // BGV add inplace

        auto message1 = randomVector(slot_size, data_bound);
        Plaintext plaintext1;
        encoder.encode(message1, plaintext1);
        Ciphertext ciphertext1;
        encryptor.encrypt(plaintext1, ciphertext1);
        CiphertextCuda c_ciphertext1(ciphertext1);
        
        auto message2 = randomVector(slot_size, data_bound);
        Plaintext plaintext2;
        encoder.encode(message2, plaintext2);
        Ciphertext ciphertext2;
        encryptor.encrypt(plaintext2, ciphertext2);
        CiphertextCuda c_ciphertext2(ciphertext2);

        c_evaluator.addInplace(c_ciphertext1, c_ciphertext2);
        
        Ciphertext result = c_ciphertext1.cpu();
        Plaintext decrypted;
        decryptor.decrypt(result, decrypted);
        
        vector<int64_t> output(slot_size);
        encoder.decode(decrypted, output);
        
        auto mexpect = addVector(message1, message2);
        printVector(mexpect, true);
        printVector(output, true);

    }

    if (false) { // BGV negate inplace
        auto message = randomVector(slot_size, data_bound);
        CiphertextCuda c1 = encryptCuda(context, encoder, encryptor, message);
        c_evaluator.negateInplace(c1);
        auto decrypted = decryptCuda(encoder, decryptor, c1, slot_size);
        printVector(message, false);
        printVector(decrypted, false);
    }


    if (false) { // BGV multiply inplace
        auto message1 = randomVector(slot_size, data_bound);
        auto message2 = randomVector(slot_size, data_bound);
        printf("before encrypt\n");
        auto cipher1 = ENCRYPT(message1);
        auto cipher2 = ENCRYPT(message2);
        printf("after encrypt\n");
        c_evaluator.multiplyInplace(cipher1, cipher2);
        printf("after mul\n");
        auto mmul = DECRYPT(cipher1);
        auto mexpect = multiplyVector(message1, message2);
        printVector(message1, false);
        printVector(message2, false);
        printVector(mmul, false);
        printVector(mexpect, false);
    }

    if (true) { // BGV square inplace
        auto message = randomVector(slot_size, data_bound);
        CiphertextCuda c1 = encryptCuda(context, encoder, encryptor, message);
        c_evaluator.squareInplace(c1);
        auto decrypted = decryptCuda(encoder, decryptor, c1, slot_size);
        printVector(multiplyVector(message, message), false);
        printVector(decrypted, false);
    }

}

int main() {
    KernelProvider::initialize();
    test_ckks();
    return 0;
}